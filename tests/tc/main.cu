// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <deque>
#include <hipsparse.h>
#include <moderngpu.cuh>

#include <util.cuh>
#include <bfs.cuh>
//#include <spmspvMM.cuh>

#include <testBfs.cpp>
#include <string.h>

void runBfs(int argc, char**argv) { 
    int m, n, edge;
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype edge value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    if( parseArgs( argc, argv, source, device, delta )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);
    
    // 2. Reads in number of edges, number of nodes
    readEdge( m, n, edge, stdin );
    printf("Graph has %d nodes, %d edges\n", m, edge);

    // 3. Allocate memory depending on how many edges are present
    typeVal *h_csrValA;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA;
    int *h_bfsResult, *h_bfsResultCPU;

    h_csrValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_bfsResult = (int*)malloc((m)*sizeof(int));
    h_bfsResultCPU = (int*)malloc((m)*sizeof(int));

    // 4. Read in graph from .mtx file
    readMtx<typeVal>( edge, h_csrColIndA, h_cooRowIndA, h_csrValA );
    print_array( h_cooRowIndA, m );

    // 5. Allocate GPU memory
    typeVal *d_csrValA;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
    typeVal *d_cscValA;
    int *d_cscRowIndA, *d_cscColPtrA;
    int *d_bfsResult;
    hipMalloc(&d_bfsResult, m*sizeof(int));

    hipMalloc(&d_csrValA, edge*sizeof(typeVal));
    hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, edge*sizeof(int));
    hipMalloc(&d_cooRowIndA, edge*sizeof(int));

    hipMalloc(&d_cscValA, edge*sizeof(typeVal));
    hipMalloc(&d_cscRowIndA, edge*sizeof(int));
    hipMalloc(&d_cscColPtrA, (m+1)*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_csrValA, h_csrValA, (edge)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cooRowIndA, h_cooRowIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);

    // 7. Run COO -> CSR kernel
    coo2csr( d_cooRowIndA, edge, m, d_csrRowPtrA );

    // 8. Run BFS on CPU. Need data in CSR form first.
    hipMemcpy(h_csrRowPtrA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    int depth = 1000;
    depth = bfsCPU( source, m, h_csrRowPtrA, h_csrColIndA, h_bfsResultCPU, depth );
    print_end_interesting(h_bfsResultCPU, m);

    // Make two GPU timers
    GpuTimer gpu_timer;
    GpuTimer gpu_timer2;
    float elapsed = 0.0f;
    float elapsed2 = 0.0f;
    gpu_timer.Start();

    // 9. Run CSR -> CSC kernel
    csr2csc<typeVal>( m, edge, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_cscValA, d_cscRowIndA, d_cscColPtrA );
    gpu_timer.Stop();
    gpu_timer2.Start();

    // 10. Run BFS kernel on GPU
    //bfs<typeVal>( source, edge, m, d_csrValA, d_cscColPtrA, d_cscRowIndA, d_bfsResult, depth, *context );
    bfs<typeVal>( source, edge, m, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_bfsResult, depth, *context );

    // 10. Run BFS kernel on GPU
    //spmspvBfs( source, edge, m, h_csrRowPtrA, d_csrRowPtrA, d_csrColIndA, d_bfsResult, depth, *context); 
    //bfs( 0, edge, m, d_cscColPtrA, d_cscRowIndA, d_bfsResult, depth, *context);
    gpu_timer2.Stop();
    elapsed += gpu_timer.ElapsedMillis();
    elapsed2 += gpu_timer2.ElapsedMillis();

    printf("CSR->CSC finished in %f msec. performed %d iterations\n", elapsed, depth-1);
    //printf("GPU BFS finished in %f msec. not including transpose\n", elapsed2);

    hipMemcpy(h_csrColIndA, d_csrColIndA, edge*sizeof(int), hipMemcpyDeviceToHost);
    print_array(h_csrColIndA, m);

    // Compare with CPU BFS for errors
    hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    verify( m, h_bfsResult, h_bfsResultCPU );
    print_array(h_bfsResult, m);

    // Compare with SpMV for errors
    //bfs( 0, edge, m, d_cscColPtrA, d_cscRowIndA, d_bfsResult, depth, *context);
    //hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    //verify( m, h_bfsResult, h_bfsResultCPU );
    //print_array(h_bfsResult, m);
    
    hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_cooRowIndA);

    hipFree(d_cscValA);
    hipFree(d_cscRowIndA);
    hipFree(d_cscColPtrA);
    hipFree(d_bfsResult);

    free(h_csrValA);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooRowIndA);
    free(h_bfsResult);
    free(h_bfsResultCPU);

    //free(h_cscValA);
    //free(h_cscRowIndA);
    //free(h_cscColPtrA);
}

int main(int argc, char**argv) {
    runBfs(argc, argv);
}    
