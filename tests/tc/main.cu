// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <deque>
#include <hipsparse.h>
#include <moderngpu.cuh>

#include <util.cuh>
#include <tc.cuh>

#include <testBfs.cpp>
#include <string.h>


int countDiag( const int edge, const int *h_cscRowIndA, const int* h_cooColIndA ) {
    int count = 0;
    for( int i=0; i<edge; i++ )
        if( h_cscRowIndA[i]==h_cooColIndA[i] )
            count++;
    return count;
}

template< typename T >
void buildVal( const int edge, T *h_cscValA ) {
    for( int i=0; i<edge; i++ )
        h_cscValA[i] = 1.0;
}

template< typename T >
void buildLower( const int m, const int edge_B, const int *h_cscColPtrA, const int *h_cscRowIndA, const T *h_cscValA, int *h_cscColPtrB, int *h_cscRowIndB, T *h_cscValB, bool lower=true ) {

    int count = 0;
    h_cscColPtrB[0] = count;
    for( int i=0; i<m; i++ ) {
        for( int j=h_cscColPtrA[i]; j<h_cscColPtrA[i+1]; j++ ) {
            if( (lower==true && h_cscRowIndA[j] > i) || (lower==false && h_cscRowIndA[j] < i) ) {
                printf("%d %d %d\n", i, j, count);
                h_cscRowIndB[count] = h_cscRowIndA[j];
                h_cscValB[count] = h_cscValA[j];
                count++;
            }
        }
        h_cscColPtrB[i+1] = count;
    }
}

void runBfs(int argc, char**argv) { 
    int m, n, edge;
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype edge value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    bool undirected = false;
    if( parseArgs( argc, argv, source, device, delta, undirected )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);
    
    // 2. Reads in number of edges, number of nodes
    readEdge( m, n, edge, stdin );
    printf("Graph has %d nodes, %d edges\n", m, edge);
    //if( undirected ) edge=2*edge;

    // 3. Allocate memory depending on how many edges are present
    typeVal *h_cscValA, *h_cooValA;
    //int *h_csrRowPtrA, *h_csrColIndA;
    int *h_cscRowIndA, *h_cscColPtrA;
    int *h_cooRowIndA, *h_cooColIndA;
    int *h_bfsResult, *h_bfsResultCPU;

    //h_csrValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_cscValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_cooValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    //h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    //h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_cscColPtrA = (int*)malloc((m+1)*sizeof(int));
    h_cscRowIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_cooColIndA = (int*)malloc(edge*sizeof(int));
    h_bfsResult = (int*)malloc((m)*sizeof(int));
    h_bfsResultCPU = (int*)malloc((m)*sizeof(int));

    // 4. Read in graph from .mtx file
    // We are actually doing readMtx<typeVal>( edge, h_cooColIndA, h_cscRowIndA, h_cscValA );
    CpuTimer cpu_timerRead;
    CpuTimer cpu_timerMake;
    CpuTimer cpu_timerBuild;
    printf("Old edge #: %d\n", edge);
    cpu_timerRead.Start();
    readMtx<typeVal>( edge/2, h_cooColIndA, h_cooRowIndA, h_cooValA );
    cpu_timerRead.Stop();
    cpu_timerMake.Start();
    edge = makeSymmetric( edge, h_cooColIndA, h_cooRowIndA, h_cooValA );
    cpu_timerMake.Stop();
    printf("\nUndirected graph has %d nodes, %d edges\n", m, edge);
    cpu_timerBuild.Start();
    buildMatrix<typeVal>( h_cscColPtrA, h_cscRowIndA, h_cscValA, m, edge, h_cooRowIndA, h_cooColIndA, h_cooValA );
    cpu_timerBuild.Stop();
    float elapsedRead = cpu_timerRead.ElapsedMillis();
    float elapsedMake = cpu_timerMake.ElapsedMillis();
    float elapsedBuild= cpu_timerBuild.ElapsedMillis();
    printf("readMtx: %f ms\n", elapsedRead);
    printf("makeSym: %f ms\n", elapsedMake);
    printf("buildMat: %f ms\n", elapsedBuild);

    // 4b. Count diagonal
    int diag = countDiag( edge, h_cooRowIndA, h_cooColIndA ); 
    int edge_B = (edge-diag)/2;
    int edge_C = edge_B;
    printf("Number of elements on diagonal: %d\n", diag);
    printf("Number of elements on L: %d\n", edge_B);

    // 4c. Allocate memory to second and third matrices
    int m_B = m;
    int m_C = m;

    typeVal *h_cscValB, *h_cscValC;
    int *h_cscRowIndB, *h_cscColPtrB;
    int *h_cscRowIndC, *h_cscColPtrC;
    h_cscValB = (typeVal*)malloc(edge_B*sizeof(typeVal));
    h_cscValC = (typeVal*)malloc(edge_C*sizeof(typeVal));
    h_cscRowIndB = (int*)malloc(edge_B*sizeof(int));
    h_cscRowIndC = (int*)malloc(edge_C*sizeof(int));
    h_cscColPtrB = (int*)malloc(m_B*sizeof(int));
    h_cscColPtrC = (int*)malloc(m_C*sizeof(int));

    buildVal( edge, h_cscValA );
    buildLower( m, edge_B, h_cscColPtrA, h_cscRowIndA, h_cscValA, h_cscColPtrB, h_cscRowIndB, h_cscValB );
    //print_matrix( h_cscValA, h_cscColPtrA, h_cscRowIndA, m );
    //print_matrix( h_cscValB, h_cscColPtrB, h_cscRowIndB, m );
    buildLower( m, edge_C, h_cscColPtrA, h_cscRowIndA, h_cscValA, h_cscColPtrC, h_cscRowIndC, h_cscValC, false );
    //print_matrix( h_cscValC, h_cscColPtrC, h_cscRowIndC, m );

    // 5. Allocate GPU memory
    typeVal *d_cscValA, *d_cscValB, *d_cscValC;
    typeVal *d_csrValA;
    int *d_csrRowPtrA, *d_csrColIndA;
    int *d_cscRowIndA, *d_cscColPtrA;
    int *d_cscRowIndB, *d_cscColPtrB;
    int *d_cscRowIndC, *d_cscColPtrC;
    int *d_cooColIndA;
    int *d_bfsResult;
    hipMalloc(&d_bfsResult, m*sizeof(int));

    //hipMalloc(&d_csrValA, edge*sizeof(typeVal));
    //hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    //hipMalloc(&d_csrColIndA, edge*sizeof(int));
    //hipMalloc(&d_cooRowIndA, edge*sizeof(int));
    //hipMalloc(&d_cooColIndA, edge*sizeof(int));
    hipMalloc(&d_cscValA, edge*sizeof(typeVal));
    hipMalloc(&d_cscRowIndA, edge*sizeof(int));
    hipMalloc(&d_cscColPtrA, (m+1)*sizeof(int));

    // 5b GPU memory for matrices B and C
    hipMalloc(&d_cscValB, edge_B*sizeof(typeVal));
    hipMalloc(&d_cscRowIndB, edge_B*sizeof(int));
    hipMalloc(&d_cscColPtrB, (m_B+1)*sizeof(int));
    hipMalloc(&d_cscValC, edge_C*sizeof(typeVal)); // Allocate C in mXm
    hipMalloc(&d_cscRowIndC, edge_C*sizeof(int));
    hipMalloc(&d_cscColPtrC, (m_C+1)*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_cscValA, h_cscValA, (edge)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_cscRowIndA, h_cscRowIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cscColPtrA, h_cscColPtrA, (m+1)*sizeof(int),hipMemcpyHostToDevice);

    // 6b Copy data from host to device for matrices B and C
    hipMemcpy(d_cscValB, h_cscValB, (edge_B)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_cscRowIndB, h_cscRowIndB, (edge_B)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cscColPtrB, h_cscColPtrB, (m+1)*sizeof(int),hipMemcpyHostToDevice);

    hipMemcpy(d_cscValC, h_cscValC, (edge_C)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_cscRowIndC, h_cscRowIndC, (edge_C)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cscColPtrC, h_cscColPtrC, (m+1)*sizeof(int),hipMemcpyHostToDevice);

    // 7. [insert CPU verification code here] 

    // 8. Make GPU timers
    GpuTimer gpu_timer;
    float elapsed = 0.0f;
    gpu_timer.Start();

    // 9. Initialize product matrix D
    

    // 10. Print two matrices

    // 11. Run spgemm
    //mXm<typeVal>( edge, m, d_cscValB, d_cscColPtrB, d_cscRowIndB, d_cscValC, h_cscColPtrC, d_cscColPtrC, d_cscRowIndC, d_cscValD, d_cscColPtrD, d_cscRowIndD, *context);
    edge_D = spgemm<typeVal>( edge, m, d_cscValB, d_cscColPtrB, d_cscRowIndB, d_cscValC, d_cscColPtrC, d_cscRowIndC, d_cscValD, d_cscColPtrD, d_cscRowIndD );

    gpu_timer.Stop();
    elapsed += gpu_timer.ElapsedMillis();

    printf("CSR->CSC finished in %f msec. performed %d iterations\n", elapsed, depth-1);
    printf("GPU BFS finished in %f msec. not including transpose\n", elapsed2);

    hipMemcpy(h_cscRowIndA, d_cscRowIndA, edge*sizeof(int), hipMemcpyDeviceToHost);
    print_array(h_cscRowIndA, m);

    // 11. Compare with CPU BFS for errors
    hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    verify( m, h_bfsResult, h_bfsResultCPU );
    //print_array(h_bfsResult, m);
    h_cscValC = (typeVal*)malloc(edge_C*sizeof(typeVal));
    h_cscRowIndC = (int*)malloc(edge_C*sizeof(int));
    hipMemcpy(h_cscRowIndC, d_cscRowIndC, edge_C*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_cscColPtrC, d_cscColPtrC, (m+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_cscValC, d_cscValC, edge_C*sizeof(float), hipMemcpyDeviceToHost);
    printf("Matrix C: %dx%d with %d nnz\n", m, m, edge_C);
    print_matrix( h_cscValC, h_cscColPtrC, h_cscRowIndC, m );
}

int main(int argc, char**argv) {
    runBfs(argc, argv);
}    
