// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsparse.h>
#include <moderngpu.cuh>

#include <util.cuh>
#include <spmspvMis.cuh>

#include <string.h>


// A simple CPU-based reference MIS ranking implementation
template<typename VertexId>
int SimpleReferenceMis(
    const VertexId m, const VertexId *h_rowPtrA, const VertexId *h_colIndA,
    VertexId                                *source_path,
    VertexId                                src)
{
    //initialize distances
    for (VertexId i = 0; i < m; ++i) {
        source_path[i] = -1;
    }
    source_path[src] = 1;
    int edges_begin = h_rowPtrA[src];
    int edges_end = h_rowPtrA[src + 1];

    for( int edge=edges_begin; edge<edges_end; edge++ ) {
        VertexId neighbor = h_colIndA[edge];

        if (source_path[neighbor] == -1)
            source_path[neighbor] = 0;
    }
    
    VertexId search_depth = 1;

    //
    //Perform MIS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
   
    for( VertexId i=0; i<m; i++ ) {
        if( source_path[i]==-1 ) {
            source_path[i] = 1;
            
            // Locate adjacency list 
            edges_begin = h_rowPtrA[i];
            edges_end = h_rowPtrA[i + 1];

            /*for( int edge=edges_begin; edge<edges_end; edge++ ) {
                VertexId neighbor = h_colIndA[edge];
                if( neighbor==i ) {
                    flag = 1;
                    source_path[i] = 0;
                    break;
                }
            }
            if( flag!=1 )*/
                for( int edge=edges_begin; edge<edges_end; edge++ ) {
                    VertexId neighbor = h_colIndA[edge];

                    if( source_path[neighbor]==-1 )
                        source_path[neighbor] = 0;
             }
        }
    }
 
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU MIS finished in %lf msec. Search depth is: %d\n", elapsed, search_depth);

    return search_depth;
}

int misCPU( const int src, const int m, const int *h_rowPtr, const int *h_colInd, int *h_misResultCPU ) {

    typedef int VertexId; // Use as the node identifier type

    int depth = SimpleReferenceMis<VertexId>(
        m, h_rowPtr, h_colInd,
        h_misResultCPU,
        src);

    //print_array(h_misResultCPU, m);
    return depth;
}

void verifyMis( const int edge, const int m, const int *d_misResultCPU, const int *d_csrRowPtr, const int *d_csrColInd, int *d_result, mgpu::CudaContext& context ) {
    spmspvCsr<int>( d_misResultCPU, edge, m, d_csrRowPtr, d_csrColInd, d_result, context );
}

void fillUniform( float *d_A, int edge ) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator( &prng, HIPRAND_RNG_PSEUDO_DEFAULT );

    hiprandGenerateUniform( prng, d_A, edge );
}

void runMis(int argc, char**argv) { 
    int m, n, edge;
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype edge value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    bool undirected = false;
    if( parseArgs( argc, argv, source, device, delta, undirected )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);
    
    // 2. Reads in number of edges, number of nodes
    readEdge( m, n, edge, stdin );
    printf("Graph has %d nodes, %d edges\n", m, edge);

    // Double # of edges because symmetric/undirected
    edge *= 2;

    // 3. Allocate memory depending on how many edges are present
    typeVal *h_randVec;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA;
    int *h_misResult, *h_misResultCPU;

    h_randVec    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_misResult = (int*)malloc((m)*sizeof(int));
    h_misResultCPU = (int*)malloc((m)*sizeof(int));

    // 4. Read in graph from .mtx file
    readMtx<typeVal>( edge/2, h_csrColIndA, h_cooRowIndA, h_randVec );
    //print_array( h_cooRowIndA, 40 );
    edge = makeSymmetric( edge, h_csrColIndA, h_cooRowIndA, h_randVec );
    printf("Undirected graph has %d edges\n", edge);

    // 5. Allocate GPU memory
    typeVal *d_randVec;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
    int *d_misResult;
    hipMalloc(&d_misResult, m*sizeof(int));

    hipMalloc(&d_randVec, edge*sizeof(typeVal));
    hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, edge*sizeof(int));
    hipMalloc(&d_cooRowIndA, edge*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_randVec, h_randVec, (edge)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cooRowIndA, h_cooRowIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);

    // 7. Run COO -> CSR kernel
    coo2csr( d_cooRowIndA, edge, m, d_csrRowPtrA );

    // 8. Run MIS on CPU. Need data in CSR form first.
    hipMemcpy(h_csrRowPtrA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    misCPU( source, m, h_csrRowPtrA, h_csrColIndA, h_misResultCPU );
    print_end_interesting(h_misResultCPU, m);

    // 9. Verify CPU-MIS by running BFS 1x on GPU.
    hipMemcpy( d_misResult, h_misResultCPU, m*sizeof(int), hipMemcpyHostToDevice );
    verifyMis( edge, m, d_misResult, d_csrRowPtrA, d_csrColIndA, d_cooRowIndA, *context );
    hipMemcpy( h_misResult, d_cooRowIndA, m*sizeof(int), hipMemcpyDeviceToHost );
    unverify( m, h_misResult, h_misResultCPU );

    // Make two GPU timers
    GpuTimer gpu_timer;
    GpuTimer gpu_timer2;
    float elapsed = 0.0f;
    float elapsed2 = 0.0f;
    gpu_timer.Start();

    // 9. Generate random numbers
    //mis( i, edge, m, d_randVec, d_csrRowPtrA, d_csrColIndA, d_misResult, 5 );
    fillUniform( d_randVec, m);
    hipMemcpy( h_randVec, d_randVec, m*sizeof(typeVal), hipMemcpyDeviceToHost );
    //print_array( h_randVec, 40);

    // 10. Run MIS kernel on GPU
    spmspvMis( edge, m, h_csrRowPtrA, d_csrRowPtrA, d_csrColIndA, d_randVec, d_misResult, delta, *context); 
    //mis( edge, m, d_csrRowPtrA, d_csrColIndA, d_misResult, delta, *context);
    gpu_timer.Stop();
    elapsed += gpu_timer.ElapsedMillis();
    elapsed2 += gpu_timer2.ElapsedMillis();

    printf("using step-size %f in %f ms\n", delta, elapsed);
    //printf("GPU MIS finished in %f msec. not including transpose\n", elapsed2);

    //hipMemcpy(h_csrColIndA, d_csrColIndA, edge*sizeof(int), hipMemcpyDeviceToHost);
    //print_array(h_csrColIndA, m);

    // 11. Verify GPU-MIS by running BFS 1x on GPU
    hipMemcpy( h_misResultCPU, d_misResult, m*sizeof(int), hipMemcpyDeviceToHost );
    verifyMis( edge, m, d_misResult, d_csrRowPtrA, d_csrColIndA, d_cooRowIndA, *context );
    hipMemcpy( h_misResult, d_cooRowIndA, m*sizeof(int), hipMemcpyDeviceToHost );
    unverify( m, h_misResult, h_misResultCPU );

    // Compare with SpMV for errors
    //cuspMis( 0, edge, m, d_csrRowPtrA, d_csrColIndA, d_misResult, depth, *context);
    //hipMemcpy(h_misResult,d_misResult,m*sizeof(int),hipMemcpyDeviceToHost);
    //verify( m, h_misResult, h_misResultCPU );
    //print_array(h_misResult, m);
    
    hipFree(d_randVec);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_cooRowIndA);
    hipFree(d_misResult);

    free(h_randVec);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooRowIndA);
    free(h_misResult);
    free(h_misResultCPU);

}

int main(int argc, char**argv) {
    runMis(argc, argv);
}    
