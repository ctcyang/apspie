// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipsparse.h>
#include <moderngpu.cuh>

#include <util.cuh>
#include <spmspvMis.cuh>

#include <string.h>


// A simple CPU-based reference MIS ranking implementation
template<typename VertexId>
int SimpleReferenceMis(
    const VertexId m, const VertexId *h_rowPtrA, const VertexId *h_colIndA,
    VertexId                                *source_path,
    VertexId                                src)
{
    //initialize distances
    for (VertexId i = 0; i < m; ++i) {
        source_path[i] = -1;
    }
    source_path[src] = 1;
    int edges_begin = h_rowPtrA[src];
    int edges_end = h_rowPtrA[src + 1];

    for( int edge=edges_begin; edge<edges_end; edge++ ) {
        VertexId neighbor = h_colIndA[edge];

        if (source_path[neighbor] == -1)
            source_path[neighbor] = 0;
    }
    
    VertexId search_depth = 1;

    //
    //Perform MIS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
   
    for( VertexId i=0; i<m; i++ ) {
        if( source_path[i]==-1 ) {
            source_path[i] = 1;
            
            // Locate adjacency list 
            edges_begin = h_rowPtrA[i];
            edges_end = h_rowPtrA[i + 1];

            /*for( int edge=edges_begin; edge<edges_end; edge++ ) {
                VertexId neighbor = h_colIndA[edge];
                if( neighbor==i ) {
                    flag = 1;
                    source_path[i] = 0;
                    break;
                }
            }
            if( flag!=1 )*/
                for( int edge=edges_begin; edge<edges_end; edge++ ) {
                    VertexId neighbor = h_colIndA[edge];

                    if( source_path[neighbor]==-1 )
                        source_path[neighbor] = 0;
             }
        }
    }
 
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU MIS finished in %lf msec. Search depth is: %d\n", elapsed, search_depth);

    return search_depth;
}

int misCPU( const int src, const int m, const int *h_rowPtr, const int *h_colInd, int *h_misResultCPU ) {

    typedef int VertexId; // Use as the node identifier type

    int depth = SimpleReferenceMis<VertexId>(
        m, h_rowPtr, h_colInd,
        h_misResultCPU,
        src);

    //print_array(h_misResultCPU, m);
    return depth;
}

void verifyMis( const int edge, const int m, const int *d_misResultCPU, const int *d_csrRowPtr, const int *d_csrColInd, int *d_result, mgpu::CudaContext& context ) {
    spmspvCsr<int>( d_misResultCPU, edge, m, d_csrRowPtr, d_csrColInd, d_result, context );
}

void fillUniform( float *d_A, int edge ) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator( &prng, HIPRAND_RNG_PSEUDO_DEFAULT );

    hiprandGenerateUniform( prng, d_A, edge );
}

template<typename typeVal>
int makeSymmetric( int edge, int *h_csrColIndA, int *h_cooRowIndA, typeVal *h_randVec ) {

    int realEdge = edge/2;
    
    for( int i=0; i<realEdge; i++ ) {
        h_cooRowIndA[realEdge+i] = h_csrColIndA[i];
        h_csrColIndA[realEdge+i] = h_cooRowIndA[i];
    }

    // Sort
    //struct arrayset *work = (arrayset*)malloc(edge*sizeof(arrayset));
    //work->values1 = h_cooRowIndA;
    //work->values2 = h_csrColIndA;
    struct arrayset work = { h_cooRowIndA, h_csrColIndA };
    custom_sort(&work, edge);

    int curr = h_csrColIndA[0];
    int last;
    int curr_row = h_cooRowIndA[0];
    int last_row;
    if( curr_row == curr )
        h_cooRowIndA[0] = -1;

    // Check for self-loops and repetitions, mark with -1
    for( int i=1; i<edge; i++ ) {
        last = curr;
        last_row = curr_row;
        curr = h_csrColIndA[i];
        curr_row = h_cooRowIndA[i];

        // Self-loops
        if( curr_row == curr )
            h_csrColIndA[i] = -1;
        // Repetitions
        else if( curr == last && curr_row == last_row )
            h_csrColIndA[i] = -1;
    }

    // Remove self-loops and repetitions.
    int shift = 0;
    int back = 0;
    for( int i=0; i+shift<edge; i++ ) {
        if(h_csrColIndA[i] == -1) {
            for( shift; back<=edge; shift++ ) {
                back = i+shift;
                if( h_csrColIndA[back] != -1 ) {
                    //printf("Swapping %d with %d\n", i, back ); 
                    h_csrColIndA[i] = h_csrColIndA[back];
                    h_cooRowIndA[i] = h_cooRowIndA[back];
                    h_csrColIndA[back] = -1;
                    break;
    }}}}
    return edge-shift;
}

void runMis(int argc, char**argv) { 
    int m, n, edge;
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype edge value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    if( parseArgs( argc, argv, source, device, delta )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);
    
    // 2. Reads in number of edges, number of nodes
    readEdge( m, n, edge, stdin );
    printf("Graph has %d nodes, %d edges\n", m, edge);

    // Double # of edges because symmetric/undirected
    edge *= 2;

    // 3. Allocate memory depending on how many edges are present
    typeVal *h_randVec;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA;
    int *h_misResult, *h_misResultCPU;

    h_randVec    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_misResult = (int*)malloc((m)*sizeof(int));
    h_misResultCPU = (int*)malloc((m)*sizeof(int));

    // 4. Read in graph from .mtx file
    readMtx<typeVal>( edge/2, h_csrColIndA, h_cooRowIndA, h_randVec );
    //print_array( h_cooRowIndA, 40 );
    edge = makeSymmetric( edge, h_csrColIndA, h_cooRowIndA, h_randVec );
    printf("Undirected graph has %d edges\n", edge);

    // 5. Allocate GPU memory
    typeVal *d_randVec;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
    int *d_misResult;
    hipMalloc(&d_misResult, m*sizeof(int));

    hipMalloc(&d_randVec, edge*sizeof(typeVal));
    hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, edge*sizeof(int));
    hipMalloc(&d_cooRowIndA, edge*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_randVec, h_randVec, (edge)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cooRowIndA, h_cooRowIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);

    // 7. Run COO -> CSR kernel
    coo2csr( d_cooRowIndA, edge, m, d_csrRowPtrA );

    // 8. Run MIS on CPU. Need data in CSR form first.
    hipMemcpy(h_csrRowPtrA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    misCPU( source, m, h_csrRowPtrA, h_csrColIndA, h_misResultCPU );
    print_end_interesting(h_misResultCPU, m);

    // 9. Verify CPU-MIS by running BFS 1x on GPU.
    hipMemcpy( d_misResult, h_misResultCPU, m*sizeof(int), hipMemcpyHostToDevice );
    verifyMis( edge, m, d_misResult, d_csrRowPtrA, d_csrColIndA, d_cooRowIndA, *context );
    hipMemcpy( h_misResult, d_cooRowIndA, m*sizeof(int), hipMemcpyDeviceToHost );
    unverify( m, h_misResult, h_misResultCPU );

    // Make two GPU timers
    GpuTimer gpu_timer;
    GpuTimer gpu_timer2;
    float elapsed = 0.0f;
    float elapsed2 = 0.0f;
    gpu_timer.Start();

    // 9. Generate random numbers
    //mis( i, edge, m, d_randVec, d_csrRowPtrA, d_csrColIndA, d_misResult, 5 );
    fillUniform( d_randVec, m);
    hipMemcpy( h_randVec, d_randVec, m*sizeof(typeVal), hipMemcpyDeviceToHost );
    //print_array( h_randVec, 40);

    // 10. Run MIS kernel on GPU
    spmspvMis( edge, m, h_csrRowPtrA, d_csrRowPtrA, d_csrColIndA, d_randVec, d_misResult, delta, *context); 
    //mis( edge, m, d_csrRowPtrA, d_csrColIndA, d_misResult, delta, *context);
    gpu_timer.Stop();
    elapsed += gpu_timer.ElapsedMillis();
    elapsed2 += gpu_timer2.ElapsedMillis();

    printf("using step-size %f in %f ms\n", delta, elapsed);
    //printf("GPU MIS finished in %f msec. not including transpose\n", elapsed2);

    //hipMemcpy(h_csrColIndA, d_csrColIndA, edge*sizeof(int), hipMemcpyDeviceToHost);
    //print_array(h_csrColIndA, m);

    // 11. Verify GPU-MIS by running BFS 1x on GPU
    hipMemcpy( h_misResultCPU, d_misResult, m*sizeof(int), hipMemcpyDeviceToHost );
    verifyMis( edge, m, d_misResult, d_csrRowPtrA, d_csrColIndA, d_cooRowIndA, *context );
    hipMemcpy( h_misResult, d_cooRowIndA, m*sizeof(int), hipMemcpyDeviceToHost );
    unverify( m, h_misResult, h_misResultCPU );

    // Compare with SpMV for errors
    //cuspMis( 0, edge, m, d_csrRowPtrA, d_csrColIndA, d_misResult, depth, *context);
    //hipMemcpy(h_misResult,d_misResult,m*sizeof(int),hipMemcpyDeviceToHost);
    //verify( m, h_misResult, h_misResultCPU );
    //print_array(h_misResult, m);
    
    hipFree(d_randVec);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_cooRowIndA);
    hipFree(d_misResult);

    free(h_randVec);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooRowIndA);
    free(h_misResult);
    free(h_misResultCPU);

}

int main(int argc, char**argv) {
    runMis(argc, argv);
}    
