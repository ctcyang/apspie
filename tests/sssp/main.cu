// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <queue>
#include <hipsparse.h>
#include <moderngpu.cuh>

#include <util.cuh>
//#include <sssp.cuh>

#include <string.h>

#define MARK_PREDECESSORS 0

class CompareDist {
public:
    bool operator() ( const std::pair<int, float>& lhs, const std::pair<int, float>& rhs ) const {
        return lhs.second < rhs.second;
    }
};

// A simple CPU-based reference SSSP ranking implementation
template<typename VertexId>
int SimpleReferenceSSSP(
    const VertexId m, const VertexId *h_rowPtrA, const VertexId *h_colIndA, const float *h_csrValA,
    float                                   *source_path,
    VertexId                                *predecessor,
    VertexId                                src,
    VertexId                                stop)
{
    typedef std::vector<VertexId> node_id;

    //initialize distances
    //  use -1 to represent infinity for source_path
    //                      undefined for predecessor
    for (VertexId i = 0; i < m; ++i) {
        source_path[i] = -1;
        if (MARK_PREDECESSORS)
            predecessor[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    typedef std::pair<VertexId, float> Edge;

    // Initialize queue for managing previously-discovered nodes
    std::priority_queue<std::pair<VertexId, float>, std::vector<std::pair<VertexId, float> >, CompareDist> frontier;
    //frontier.push(src);

    //
    //Perform SSSP
    //

    /*CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty()) {
        
        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;
        if( neighbor_dist > stop )
            break;

        // Locate adjacency list
        int edges_begin = h_rowPtrA[dequeued_node];
        int edges_end = h_rowPtrA[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge) {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = h_colIndA[edge];
            if (source_path[neighbor] == -1) {
                source_path[neighbor] = neighbor_dist;
                if (MARK_PREDECESSORS) {
                    predecessor[neighbor] = dequeued_node;
                }
                if (search_depth < neighbor_dist) {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    if (MARK_PREDECESSORS)
        predecessor[src] = -1;

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU SSSP finished in %lf msec. Search depth is: %d\n", elapsed, search_depth);*/

    return search_depth;
}

int ssspCPU( const int src, const int m, const int *h_rowPtrA, const int *h_colIndA, const float* h_csrValA, float *h_ssspResultCPU, const int stop ) {

    typedef int VertexId; // Use as the node identifier type

    VertexId *reference_check_preds = NULL;

    int depth = SimpleReferenceSSSP<VertexId>(
        m, h_rowPtrA, h_colIndA, h_csrValA,
        h_ssspResultCPU,
        reference_check_preds,
        src,
        stop);

    //print_array(h_ssspResultCPU, m);
    return depth;
}

void runSSSP(int argc, char**argv) { 
    int m, n, edge;
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype edge value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    if( parseArgs( argc, argv, source, device, delta )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);
    
    // 2. Reads in number of edges, number of nodes
    readEdge( m, n, edge, stdin );
    printf("Graph has %d nodes, %d edges\n", m, edge);

    // 3. Allocate memory depending on how many edges are present
    typeVal *h_csrValA;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA;
    float *h_ssspResult, *h_ssspResultCPU;

    h_csrValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_ssspResult = (float*)malloc((m)*sizeof(float));
    h_ssspResultCPU = (float*)malloc((m)*sizeof(float));

    // 4. Read in graph from .mtx file
    readMtx<typeVal>( edge, h_csrColIndA, h_cooRowIndA, h_csrValA );
    print_array( h_csrRowPtrA, m );

    // 5. Allocate GPU memory
    typeVal *d_csrValA;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
    typeVal *d_cscValA;
    int *d_cscRowIndA, *d_cscColPtrA;
    float *d_ssspResult;
    hipMalloc(&d_ssspResult, m*sizeof(float));

    hipMalloc(&d_csrValA, edge*sizeof(typeVal));
    hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, edge*sizeof(int));
    hipMalloc(&d_cooRowIndA, edge*sizeof(int));

    hipMalloc(&d_cscValA, edge*sizeof(typeVal));
    hipMalloc(&d_cscRowIndA, edge*sizeof(int));
    hipMalloc(&d_cscColPtrA, (m+1)*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_csrValA, h_csrValA, (edge)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cooRowIndA, h_cooRowIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);

    // 7. Run COO -> CSR kernel
    coo2csr( d_cooRowIndA, edge, m, d_csrRowPtrA );

    // 8. Run SSSP on CPU. Need data in CSR form first.
    hipMemcpy(h_csrRowPtrA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    int depth = 1000;
    depth = ssspCPU( source, m, h_csrRowPtrA, h_csrColIndA, h_csrValA, h_ssspResultCPU, depth );
    print_end_interesting(h_ssspResultCPU, m);

    // Make two GPU timers
    /*GpuTimer gpu_timer;
    GpuTimer gpu_timer2;
    float elapsed = 0.0f;
    float elapsed2 = 0.0f;
    gpu_timer.Start();

    // 9. Run CSR -> CSC kernel
    csr2csc<typeVal>( m, edge, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_cscValA, d_cscRowIndA, d_cscColPtrA );
    gpu_timer.Stop();
    gpu_timer2.Start();

    // 10. Run SSSP kernel on GPU
    //sssp<typeVal>( source, edge, m, d_csrValA, d_cscColPtrA, d_cscRowIndA, d_ssspResult, depth, *context );
    sssp<typeVal>( source, edge, m, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_ssspResult, depth, *context );

    gpu_timer2.Stop();
    elapsed += gpu_timer.ElapsedMillis();
    elapsed2 += gpu_timer2.ElapsedMillis();

    printf("CSR->CSC finished in %f msec. performed %d iterations\n", elapsed, depth-1);
    //printf("GPU SSSP finished in %f msec. not including transpose\n", elapsed2);

    hipMemcpy(h_csrColIndA, d_csrColIndA, edge*sizeof(int), hipMemcpyDeviceToHost);
    print_array(h_csrColIndA, m);

    // Compare with CPU SSSP for errors
    hipMemcpy(h_ssspResult,d_ssspResult,m*sizeof(float),hipMemcpyDeviceToHost);
    verify( m, h_ssspResult, h_ssspResultCPU );
    print_array(h_ssspResult, m);

    // Compare with SpMV for errors
    //bfs( 0, edge, m, d_cscColPtrA, d_cscRowIndA, d_bfsResult, depth, *context);
    //hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    //verify( m, h_bfsResult, h_bfsResultCPU );
    //print_array(h_bfsResult, m);
    
    hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_cooRowIndA);

    hipFree(d_cscValA);
    hipFree(d_cscRowIndA);
    hipFree(d_cscColPtrA);
    hipFree(d_ssspResult);

    free(h_csrValA);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooRowIndA);
    free(h_ssspResult);
    free(h_ssspResultCPU);

    //free(h_cscValA);
    //free(h_cscRowIndA);
    //free(h_cscColPtrA);*/
}

int main(int argc, char**argv) {
    runSSSP(argc, argv);
}    
