// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <deque>
#include <hipsparse.h>

#include <util.cuh>
#include <bfs.cuh>
#include <spmspvBfs.cuh>

#include <string.h>

#define MARK_PREDECESSORS 0

// A simple CPU-based reference BFS ranking implementation
template<typename VertexId>
int SimpleReferenceBfs(
    const VertexId m, const VertexId *h_rowPtrA, const VertexId *h_colIndA,
    VertexId                                *source_path,
    VertexId                                *predecessor,
    VertexId                                src,
    VertexId                                stop)
{
    //initialize distances
    for (VertexId i = 0; i < m; ++i) {
        source_path[i] = -1;
        if (MARK_PREDECESSORS)
            predecessor[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    //
    //Perform BFS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty()) {
        
        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;
        if( neighbor_dist > stop )
            break;

        // Locate adjacency list
        int edges_begin = h_rowPtrA[dequeued_node];
        int edges_end = h_rowPtrA[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge) {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = h_colIndA[edge];
            if (source_path[neighbor] == -1) {
                source_path[neighbor] = neighbor_dist;
                if (MARK_PREDECESSORS) {
                    predecessor[neighbor] = dequeued_node;
                }
                if (search_depth < neighbor_dist) {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    if (MARK_PREDECESSORS)
        predecessor[src] = -1;

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU BFS finished in %lf msec. Search depth is: %d\n", elapsed, search_depth);

    return search_depth;
}

int bfsCPU( const int src, const int m, const int *h_rowPtrA, const int *h_colIndA, int *h_bfsResultCPU, const int stop ) {

    typedef int VertexId; // Use as the node identifier type

    VertexId *reference_check_preds = NULL;

    int depth = SimpleReferenceBfs<VertexId>(
        m, h_rowPtrA, h_colIndA,
        h_bfsResultCPU,
        reference_check_preds,
        src,
        stop);

    //print_array(h_bfsResultCPU, m);
    return depth;
}

void runBfs(int argc, char**argv) { 
    int m, n, edge;
    ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype edge value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    if( parseArgs( argc, argv, source, device, delta )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);
    
    // 2. Reads in number of edges, number of nodes
    readEdge( m, n, edge, stdin );
    printf("Graph has %d nodes, %d edges\n", m, edge);

    // 3. Allocate memory depending on how many edges are present
    typeVal *h_csrValA;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA;
    int *h_bfsResult, *h_bfsResultCPU;

    h_csrValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_bfsResult = (int*)malloc((m)*sizeof(int));
    h_bfsResultCPU = (int*)malloc((m)*sizeof(int));

    // 4. Read in graph from .mtx file
    readMtx<typeVal>( edge, h_csrColIndA, h_cooRowIndA, h_csrValA );
    print_array( h_csrRowPtrA, m );

    // 5. Allocate GPU memory
    typeVal *d_csrValA;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
    typeVal *d_cscValA;
    int *d_cscRowIndA, *d_cscColPtrA;
    int *d_bfsResult;
    hipMalloc(&d_bfsResult, m*sizeof(int));

    hipMalloc(&d_csrValA, edge*sizeof(typeVal));
    hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, edge*sizeof(int));
    hipMalloc(&d_cooRowIndA, edge*sizeof(int));

    hipMalloc(&d_cscValA, edge*sizeof(typeVal));
    hipMalloc(&d_cscRowIndA, edge*sizeof(int));
    hipMalloc(&d_cscColPtrA, (m+1)*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_csrValA, h_csrValA, (edge)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cooRowIndA, h_cooRowIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);

    // 7. Run COO -> CSR kernel
    coo2csr( d_cooRowIndA, edge, m, d_csrRowPtrA );

    // 8. Run BFS on CPU. Need data in CSR form first.
    hipMemcpy(h_csrRowPtrA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    int depth = 1000;
    depth = bfsCPU( source, m, h_csrRowPtrA, h_csrColIndA, h_bfsResultCPU, depth );
    print_end_interesting(h_bfsResultCPU, m);

    // Make two GPU timers
    GpuTimer gpu_timer;
    GpuTimer gpu_timer2;
    float elapsed = 0.0f;
    float elapsed2 = 0.0f;
    gpu_timer.Start();

    // 9. Run CSR -> CSC kernel
    csr2csc<typeVal>( m, edge, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_cscValA, d_cscRowIndA, d_cscColPtrA );
    gpu_timer.Stop();
    gpu_timer2.Start();

    // 10. Run BFS kernel on GPU
    //bfs( i, edge, m, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_bfsResult, 5 );
    //bfs( 0, edge, m, d_cscValA, d_cscColPtrA, d_cscRowIndA, d_bfsResult, 5 );

    // 10. Run BFS kernel on GPU
    spmspvBfs( source, edge, m, h_csrRowPtrA, d_csrRowPtrA, d_csrColIndA, d_bfsResult, depth, *context); 
    //bfs( 0, edge, m, d_cscColPtrA, d_cscRowIndA, d_bfsResult, depth, *context);
    gpu_timer2.Stop();
    elapsed += gpu_timer.ElapsedMillis();
    elapsed2 += gpu_timer2.ElapsedMillis();

    printf("CSR->CSC finished in %f msec. performed %d iterations\n", elapsed, depth-1);
    //printf("GPU BFS finished in %f msec. not including transpose\n", elapsed2);

    hipMemcpy(h_csrColIndA, d_csrColIndA, edge*sizeof(int), hipMemcpyDeviceToHost);
    print_array(h_csrColIndA, m);

    // Compare with CPU BFS for errors
    hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    verify( m, h_bfsResult, h_bfsResultCPU );
    print_array(h_bfsResult, m);

    // Compare with SpMV for errors
    //bfs( 0, edge, m, d_cscColPtrA, d_cscRowIndA, d_bfsResult, depth, *context);
    //hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    //verify( m, h_bfsResult, h_bfsResultCPU );
    //print_array(h_bfsResult, m);
    
    hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_cooRowIndA);

    hipFree(d_cscValA);
    hipFree(d_cscRowIndA);
    hipFree(d_cscColPtrA);
    hipFree(d_bfsResult);

    free(h_csrValA);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooRowIndA);
    free(h_bfsResult);
    free(h_bfsResultCPU);

    //free(h_cscValA);
    //free(h_cscRowIndA);
    //free(h_cscColPtrA);
}

int main(int argc, char**argv) {
    runBfs(argc, argv);
}    
