#include "hip/hip_runtime.h"
// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]

#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <deque>

#include <util.cuh>
#include <bfs.cuh>

#define MARK_PREDECESSORS 0

// Verify the result
void verify( const int m, const int *h_bfsResult, const int *h_bfsResultCPU ){
    if (h_bfsResultCPU != NULL) {
        printf("Label Validity: ");
        int error_num = CompareResults(h_bfsResult, h_bfsResultCPU, m, true);
        if (error_num > 0) {
            printf("%d errors occurred.\n", error_num);
        }
    }
}

/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference BFS ranking implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  *
  * @param[in] graph Reference to the CSR graph we process on
  * @param[in] source_path Host-side vector to store CPU computed labels for each node
  * @param[in] predecessor Host-side vector to store CPU computed predecessor for each node
  * @param[in] src Source node where BFS starts
  */
template<typename VertexId>
int SimpleReferenceBfs(
    const VertexId m, const VertexId *h_rowPtrA, const VertexId *h_colIndA,
    VertexId                                *source_path,
    VertexId                                *predecessor,
    VertexId                                src,
    VertexId                                stop)
{
    //initialize distances
    for (VertexId i = 0; i < m; ++i) {
        source_path[i] = -1;
        if (MARK_PREDECESSORS)
            predecessor[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    //
    //Perform BFS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty()) {
        
        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;
        if( neighbor_dist > stop )
            break;

        // Locate adjacency list
        int edges_begin = h_rowPtrA[dequeued_node];
        int edges_end = h_rowPtrA[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge) {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = h_colIndA[edge];
            if (source_path[neighbor] == -1) {
                source_path[neighbor] = neighbor_dist;
                if (MARK_PREDECESSORS) {
                    predecessor[neighbor] = dequeued_node;
                }
                if (search_depth < neighbor_dist) {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    if (MARK_PREDECESSORS)
        predecessor[src] = -1;

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU BFS finished in %lf msec. Search depth is: %d\n", elapsed, search_depth);

    return search_depth;
}

int bfsCPU( const int src, const int m, const int *h_rowPtrA, const int *h_colIndA, int *h_bfsResultCPU, const int stop ) {

    typedef int VertexId; // Use as the node identifier type

    VertexId *reference_check_preds = NULL;

    int depth = SimpleReferenceBfs<VertexId>(
        m, h_rowPtrA, h_colIndA,
        h_bfsResultCPU,
        reference_check_preds,
        src,
        stop);

    //print_array(h_bfsResultCPU, m);
    return depth;
}

void coo2csr( const int *d_cooRowIndA, const int edge, const int m, int *d_csrRowPtrA ) {

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    hipsparseStatus_t status = hipsparseXcoo2csr(handle, d_cooRowIndA, edge, m, d_csrRowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

    switch( status ) {
        case HIPSPARSE_STATUS_SUCCESS:
            //printf("COO -> CSR conversion successful!\n");
            break;
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            printf("Error: Library not initialized.\n");
            break;
        case HIPSPARSE_STATUS_INVALID_VALUE:
            printf("Error: Invalid value for idxbase.\n");
            break;
        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            printf("Error: Failed to launch GPU.\n");
    }

    // Important: destroy handle
    hipsparseDestroy(handle);
}

void csr2csc( const int m, const int edge, const float *d_csrValA, const int *d_csrRowPtrA, const int *d_csrColIndA, float *d_cscValA, int *d_cscRowIndA, int *d_cscColPtrA ) {

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // For CUDA 4.0
    //hipsparseStatus_t status = hipsparseScsr2csc(handle, m, m, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_cscValA, d_cscRowIndA, d_cscColPtrA, 1, HIPSPARSE_INDEX_BASE_ZERO);

    // For CUDA 5.0+
    hipsparseStatus_t status = hipsparseScsr2csc(handle, m, m, edge, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_cscValA, d_cscRowIndA, d_cscColPtrA, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO);

    switch( status ) {
        case HIPSPARSE_STATUS_SUCCESS:
            //printf("Transpose successful!\n");
            break;
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            printf("Error: Library not initialized.\n");
            break;
        case HIPSPARSE_STATUS_INVALID_VALUE:
            printf("Error: Invalid parameters m, n, or nnz.\n");
            break;
        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            printf("Error: Failed to launch GPU.\n");
            break;
        case HIPSPARSE_STATUS_ALLOC_FAILED:
            printf("Error: Resources could not be allocated.\n");
            break;
        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            printf("Error: Device architecture does not support.\n");
            break;
        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            printf("Error: An internal operation failed.\n");
    }

    // Important: destroy handle
    hipsparseDestroy(handle);
}

int main(int argc, char**argv) {
    int m, n, edge;

    // Broken on graphs with more than 500k edges
    freopen(argv[1],"r",stdin);
    //freopen("log","w",stdout);
    printf("Testing %s\n", argv[1]);

    // File i/o
    bool weighted;
    int c = getchar();
    int old_c = 0;
    //printf("%d\n",c);
    while( c!=EOF ) {
        if( (old_c==10 || old_c==0) && c!=37 ) {
            ungetc(c, stdin);
            //printf("%d %d\n",old_c,c);
            break;
        }
        old_c = c;
        c=getchar();
    }
    scanf("%d %d %d", &m, &n, &edge);
    
    // Allocate memory depending on how many edges are present
    float *h_csrValA;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA;
    int *h_bfsResult, *h_bfsResultCPU;

    h_csrValA    = (float*)malloc(edge*sizeof(float));
    h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_bfsResult = (int*)malloc((m)*sizeof(int));
    h_bfsResultCPU = (int*)malloc((m)*sizeof(int));

    // Currently checks if there are fewer rows than promised
    // Could add check for edges in diagonal of adjacency matrix
    for( int j=0; j<edge; j++ ) {
        if( scanf("%d", &h_csrColIndA[j])==EOF ) {
            printf("Error: not enough rows in mtx file.\n");
            break;
        }
        scanf("%d", &h_cooRowIndA[j]);

        if( j==0 ) {
            c=getchar();
            //printf("c = %d\n",c);
        }

        if( c!=32 ) {
            h_csrValA[j]=1.0;
            if( j==0 ) weighted = false;
        } else {
            scanf("%f", &h_csrValA[j]);
        }

        h_cooRowIndA[j]--;
        h_csrColIndA[j]--;
        //printf("%d %d %d\n", h_cooRowIndA[j], h_csrColIndA[j], j);
    }
    if( weighted==true ) {
        printf("The graph is weighted: ");
        print_end(h_csrValA,edge);
    } else {
        printf("The graph is unweighted.\n");
    }

    // Allocate GPU memory
    float *d_csrValA;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
    float *d_cscValA;
    int *d_cscRowIndA, *d_cscColPtrA;
    int *d_bfsResult;

    hipMalloc(&d_bfsResult, m*sizeof(int));

    hipMalloc(&d_csrValA, edge*sizeof(float));
    hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, edge*sizeof(int));
    hipMalloc(&d_cooRowIndA, edge*sizeof(int));

    hipMalloc(&d_cscValA, edge*sizeof(float));
    hipMalloc(&d_cscRowIndA, edge*sizeof(int));
    hipMalloc(&d_cscColPtrA, (m+1)*sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_csrValA, h_csrValA, edge*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, edge*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cooRowIndA, h_cooRowIndA, edge*sizeof(int),hipMemcpyHostToDevice);

    // Run COO -> CSR kernel
    coo2csr( d_cooRowIndA, edge, m, d_csrRowPtrA );

    // Run BFS on CPU. Need data in CSR form first.
    hipMemcpy(h_csrRowPtrA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    //print_array(h_csrRowPtrA,m+1);

    int depth = 1000;
    depth = bfsCPU( 0, m, h_csrRowPtrA, h_csrColIndA, h_bfsResultCPU, depth );

    // Some testing code. To be turned into unit test.
    //int depth = 4;
    //bfsCPU( 0, m, h_csrRowPtrA, h_csrColIndA, h_bfsResultCPU, depth );
    //depth++;
    //print_end_interesting(h_bfsResultCPU, m);

    GpuTimer gpu_timer;
    GpuTimer gpu_timer2;
    float elapsed = 0.0f;
    float elapsed2 = 0.0f;
    gpu_timer.Start();

    // Run CSR -> CSC kernel
    csr2csc( m, edge, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_cscValA, d_cscRowIndA, d_cscColPtrA );
    gpu_timer2.Start();

    hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_cooRowIndA);
    hipFree(d_cscValA);

    hipMemcpy(h_csrRowPtrA, d_cscColPtrA, (m+1)*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColIndA, d_cscRowIndA, edge*sizeof(int),hipMemcpyDeviceToHost);
    hipFree(d_cscColPtrA);
    hipFree(d_cscRowIndA);

    // Allocate GPU memory for result
    float *d_spmvResult, *d_spmvSwap;
    hipMalloc(&d_spmvResult, m*sizeof(float));
    hipMalloc(&d_spmvSwap, m*sizeof(float));

    Pack pack;
    pack.copyPack( 0, m, edge, depth, h_csrRowPtrA, h_csrColIndA );

    // Run BFS kernel on GPU
    bfs( pack, d_bfsResult, d_spmvResult, d_spmvSwap );
    pack.Free(); 

    gpu_timer.Stop();
    gpu_timer2.Stop();
    elapsed += gpu_timer.ElapsedMillis();
    elapsed2 += gpu_timer2.ElapsedMillis();

    printf("GPU BFS finished in %f msec. performed %d iterations\n", elapsed, depth-1);
    printf("GPU BFS finished in %f msec. not including transpose\n", elapsed2);

    // Run check for errors
    hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    verify( m, h_bfsResult, h_bfsResultCPU );
    print_array(h_bfsResult, m);

    hipFree(d_bfsResult);

    hipFree(d_spmvResult);
    hipFree(d_spmvSwap);

    free(h_csrValA);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooRowIndA);

    free(h_bfsResult);
    free(h_bfsResultCPU);

    //free(h_cscValA);
    //free(h_cscRowIndA);
    //free(h_cscColPtrA);
}
