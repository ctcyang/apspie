// Puts everything together

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>

#define DIM 3

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

    // Destroy the handle
    hipblasDestroy(handle);
}

//Device code for naive matrix multiplication
__device__ void sgemm_helper(const float*A, const float*B, float*C, int z) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x+y*gridDim.x;

    if( z==0 )
        C[offset] = 0;
    C[offset] += B[z+y*gridDim.x] * A[x+z*gridDim.x];
}

__global__ void sgemm(const float*A, const float*B, float*C, const int m, const int n, const int k) {

    int z;

    for( z=0; z<k; z++ ) {
        sgemm_helper(A, B, C, z);
    }

}

//Calculate matrix multiplication
int gpu_sgemm(const float*A, const float*B, float*C, const int m, const int n, const int k) {;

    //Check for invalid input
    if( m<0 || n<0 || k<0 )
        return;

    dim3 grid(m,n);

    sgemm<<<grid,1>>>( A, B, C, m, n, k );
    return;
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

//Do matrix calculation before 

int main() {
    clock_t t;

    // Allocate 3 arrays on CPU
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

    // for simplicity we are going to use square arrays
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
    
    float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    // Allocate 3 arrays on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
    hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

    // Fill the arrays A and B on GPU with random numbers
    GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
    GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

    // Optionally we can copy the data back on CPU and print the arrays
    hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost);
    std::cout << "A =" << std::endl;
    print_matrix(h_A, nr_rows_A, nr_cols_A);
    std::cout << "B =" << std::endl;
    print_matrix(h_B, nr_rows_B, nr_cols_B);

    t = clock();
    std::cout << "Calculating...\n";
    // Multiply A and B on GPU
    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
    t = clock() - t;
    std::cout << "It took " << ((float)t)/CLOCKS_PER_SEC << " seconds.\n";

    // Copy (and print) the result on host memory
    hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);
    std::cout << "C =" << std::endl;
    print_matrix(h_C, nr_rows_C, nr_cols_C);

    t = clock();
    std::cout << "Calculating...\n";
    gpu_sgemm(d_A, d_B, d_C, nr_rows_A, nr_cols_B, nr_cols_A);
    t = clock() - t;
    std::cout << "It took " << ((float)t)/CLOCKS_PER_SEC << " seconds.\n";

    // Copy (and print) the result on host memory
    hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);
    std::cout << "C =" << std::endl;
    print_matrix(h_C, nr_rows_C, nr_cols_C);

    //Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);  

    // Free CPU memory
    free(h_A);
    free(h_B);
    free(h_C);    

    return 0;
}
