// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <deque>
#include <hipsparse.h>

#include <moderngpu.cuh>
#include <util.cuh>
#include <bfs.cuh>
#include <spmspvBfs.cuh>

#include <testBfs.cpp>
#include <string.h>

void runBfs(int argc, char**argv) { 
    int m, n, edge;
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype edge value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    bool undirected = false;
    if( parseArgs( argc, argv, source, device, delta, undirected )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);
    
    // 2. Reads in number of edges, number of nodes
    //    Note: Need to double # of edges in case of undirected, because this affects
    //          how much to allocate
    readEdge( m, n, edge, stdin );
    if( undirected ) 
        edge=2*edge;

    // 3. Allocate memory depending on how many edges are present
    typeVal *h_csrValA, *h_cooValA;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA, *h_cooColIndA;
    int *h_bfsResult, *h_bfsResultCPU;

    h_csrValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    h_cooValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_cooColIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_bfsResult = (int*)malloc((m)*sizeof(int));
    h_bfsResultCPU = (int*)malloc((m)*sizeof(int));

    // 4. Read in graph from .mtx file
    CpuTimer cpu_timerRead;
    CpuTimer cpu_timerMake;
    CpuTimer cpu_timerBuild;
    if( undirected ) {
        printf("Old edge #: %d\n", edge);
        cpu_timerRead.Start();
        readMtx<typeVal>( edge/2, h_cooColIndA, h_cooRowIndA, h_cooValA );
        cpu_timerRead.Stop();
        cpu_timerMake.Start();
        edge = makeSymmetric( edge, h_cooColIndA, h_cooRowIndA, h_cooValA );
        cpu_timerMake.Stop();
        printf("\nUndirected graph has %d nodes, %d edges\n", m, edge);
    } else {
        readMtx<typeVal>( edge, h_cooColIndA, h_cooRowIndA, h_cooValA );
        printf("\nDirected graph has %d nodes, %d edges\n", m, edge);
    }
    cpu_timerBuild.Start();
    buildMatrix<typeVal>( h_csrRowPtrA, h_csrColIndA, h_csrValA, m, edge, h_cooRowIndA, h_cooColIndA, h_cooValA );
    cpu_timerBuild.Stop();
    float elapsedRead = cpu_timerRead.ElapsedMillis();
    float elapsedMake = cpu_timerMake.ElapsedMillis();
    float elapsedBuild= cpu_timerBuild.ElapsedMillis();
    printf("readMtx: %f ms\n", elapsedRead);
    printf("makeSym: %f ms\n", elapsedMake);
    printf("buildMat: %f ms\n", elapsedBuild);

    /*print_array( h_cooRowIndA, m );
    print_array( h_cooColIndA, m );
    print_array( h_csrRowPtrA, m );
    print_array( h_csrColIndA, m );*/

    // 5. Allocate GPU memory
    typeVal *d_csrValA;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
    typeVal *d_cscValA;
    int *d_cscRowIndA, *d_cscColPtrA;
    int *d_bfsResult;
    hipMalloc(&d_bfsResult, m*sizeof(int));

    hipMalloc(&d_csrValA, edge*sizeof(typeVal));
    hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, edge*sizeof(int));
    //hipMalloc(&d_cooRowIndA, edge*sizeof(int));

    hipMalloc(&d_cscValA, edge*sizeof(typeVal));
    hipMalloc(&d_cscRowIndA, edge*sizeof(int));
    hipMalloc(&d_cscColPtrA, (m+1)*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_csrValA, h_csrValA, (edge)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, (m+1)*sizeof(int),hipMemcpyHostToDevice);

    // 7. Run COO -> CSR kernel
    //coo2csr( d_cooRowIndA, edge, m, d_csrRowPtrA );

    // 8. Run BFS on CPU. Need data in CSR form first.
    //hipMemcpy(h_cooRowIndA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    //verify( m, h_cooRowIndA, h_csrRowPtrA );
    //hipMemcpy(h_csrRowPtrA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    int depth = 1000;
    depth = bfsCPU( source, m, h_csrRowPtrA, h_csrColIndA, h_bfsResultCPU, depth );

    // 9. Run CSR -> CSC kernel
    //csr2csc<typeVal>( m, edge, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_cscValA, d_cscRowIndA, d_cscColPtrA );

    // 10. Run BFS kernel on GPU
    // Experiment 1: Optimized BFS using mXv (no Val array)
    //spmspvBfs( source, edge, m, h_csrRowPtrA, d_csrRowPtrA, d_csrColIndA, d_bfsResult, depth, *context); 

    // Experiment 2: Optimized BFS using mXv
    bfs( source, edge, m, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_bfsResult, depth, *context); 
    // Compare with CPU BFS for errors
    hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    verify( m, h_bfsResult, h_bfsResultCPU );
    //print_array(h_bfsResult, m);

    // Compare with SpMV for errors
    //bfs( 0, edge, m, d_cscColPtrA, d_cscRowIndA, d_bfsResult, depth, *context);
    //hipMemcpy(h_bfsResult,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    //verify( m, h_bfsResult, h_bfsResultCPU );
    //print_array(h_bfsResult, m);
    
    /*hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);

    hipFree(d_cscValA);
    hipFree(d_cscRowIndA);
    hipFree(d_cscColPtrA);
    hipFree(d_bfsResult);

    free(h_csrValA);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooValA);
    free(h_cooRowIndA);
    free(h_cooColIndA);
    free(h_bfsResult);
    free(h_bfsResultCPU);*/
}

int main(int argc, char**argv) {
    runBfs(argc, argv);
}    
