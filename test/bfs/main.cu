// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <string.h>

#include <deque>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <moderngpu.cuh>
#include <mpi.h>

#include <util.cuh>
#include <bfs.cuh>
#include <spmspvBfs.cuh>
#include <testBfs.cpp>

void runBfs(int argc, char**argv) { 
    // Initialize MPI
    MPI_Init( &argc, &argv );
    int direct, rank, size;

    int m, n, nnz;
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype nnz value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    bool undirected;
    int multi;
    if( parseArgs( argc, argv, source, device, delta, undirected, multi )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);

    // Ensure that RDMA ENABLED CUDA is set correctly
    direct = getenv("MPICH_RDMA_ENABLED_CUDA")==NULL?0:atoi(getenv ("MPICH_RDMA_ENABLED_CUDA"));
    if(direct != 1){
        printf ("MPICH_RDMA_ENABLED_CUDA not enabled!\n");
        exit (EXIT_FAILURE);
    }

    // Get MPI rank and size
    // Test whether number of MPI processes matches number passed into commandline
    MPI_Comm_rank (MPI_COMM_WORLD, &rank);
    MPI_Comm_size (MPI_COMM_WORLD, &size);
    //printf("My rank is %d\n", rank);
    if( size!=multi ) {
        printf( "Assigned node count %d != %d desired node count!\n");
        exit( EXIT_FAILURE );
    }

    // 2. Reads in number of nnzs, number of nodes
    //    Note: Need to double # of nnzs in case of undirected, because this affects
    //          how much to allocate
    //if( rank==0 ) {
    readEdge( m, n, nnz, stdin );
    if( undirected ) 
        nnz=2*nnz;

    // 3. Allocate memory depending on how many nnzs are present
    typeVal *h_csrValA, *h_cooValA;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA, *h_cooColIndA;
    int *h_bfsResult, *h_bfsResultCPU;

    h_csrValA    = (typeVal*)malloc(nnz*sizeof(typeVal));
    h_csrColIndA = (int*)malloc(nnz*sizeof(int));
    h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    h_cooValA    = (typeVal*)malloc(nnz*sizeof(typeVal));
    h_cooColIndA = (int*)malloc(nnz*sizeof(int));
    h_cooRowIndA = (int*)malloc(nnz*sizeof(int));
    h_bfsResult = (int*)malloc((m)*sizeof(int));
    h_bfsResultCPU = (int*)malloc((m)*sizeof(int));

    // 4. Read in graph from .mtx file
    CpuTimer cpu_timerRead;
    CpuTimer cpu_timerMake;
    CpuTimer cpu_timerBuild;
    if( undirected ) {
        if(rank==0)printf("Old nnz #: %d\n", nnz);
        cpu_timerRead.Start();
        readMtx<typeVal>( nnz/2, h_cooColIndA, h_cooRowIndA, h_cooValA );
        cpu_timerRead.Stop();
        cpu_timerMake.Start();
        nnz = makeSymmetric( nnz, h_cooColIndA, h_cooRowIndA, h_cooValA );
        cpu_timerMake.Stop();
        if(rank==0)printf("\nUndirected graph has %d nodes, %d nnzs\n", m, nnz);
    } else {
        readMtx<typeVal>( nnz, h_cooColIndA, h_cooRowIndA, h_cooValA );
        if(rank==0)printf("\nDirected graph has %d nodes, %d nnzs\n", m, nnz);
    }
    cpu_timerBuild.Start();
    buildMatrix<typeVal>( h_csrRowPtrA, h_csrColIndA, h_csrValA, m, nnz, h_cooRowIndA, h_cooColIndA, h_cooValA );
    cpu_timerBuild.Stop();
    float elapsedRead = cpu_timerRead.ElapsedMillis();
    float elapsedMake = cpu_timerMake.ElapsedMillis();
    float elapsedBuild= cpu_timerBuild.ElapsedMillis();
    if(rank==0) {
        printf("readMtx: %f ms\n", elapsedRead);
        printf("makeSym: %f ms\n", elapsedMake);
        printf("buildMat: %f ms\n", elapsedBuild);
    }

    /*print_array( h_cooRowIndA, m );
    print_array( h_cooColIndA, m );
    print_array( h_csrRowPtrA, m );
    print_array( h_csrColIndA, m );*/

    // 5. Allocate GPU memory
    // Multi-GPU:
    //   -Option 1:
    //   m=m/multi+1            for all
    //   
    //   nnz=same as Option 2
    //
    //   -Option 2: (not implemented yet)
    //   m=m/multi              if rank!=multi-1
    //   m=m-(multi-1)*m/multi  else
    //
    //   nnz=h_csrRowIndA[(rank+1)*m]-h_csrRowIndA[rank*m]  if rank!=multi-1
    //   nnz=nnz-h_csrRowIndA[rank*m]                      else
    int new_n, new_nnz;
    if( rank==multi-1 ) {
        new_n = m-rank*(m/multi+1)+1;
        new_nnz = nnz - h_csrRowPtrA[rank*(m/multi+1)];
    } else {
        new_n = m/multi+1;
        new_nnz = h_csrRowPtrA[(rank+1)*new_n]-h_csrRowPtrA[rank*new_n];
    }

    typeVal *d_csrValA;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;
    typeVal *d_cscValA;
    int *d_cscRowIndA, *d_cscColPtrA;
    int *d_bfsResult;
    hipMalloc(&d_bfsResult, new_n*sizeof(int));

    hipMalloc(&d_csrValA, new_nnz*sizeof(typeVal));
    hipMalloc(&d_csrRowPtrA, (new_n+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, new_nnz*sizeof(int));
    //hipMalloc(&d_cooRowIndA, new_nnz*sizeof(int));

    hipMalloc(&d_cscValA, new_nnz*sizeof(typeVal));
    hipMalloc(&d_cscRowIndA, new_nnz*sizeof(int));
    hipMalloc(&d_cscColPtrA, (new_n+1)*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_csrValA, &h_csrValA[h_csrRowPtrA[rank*(m/multi+1)]], (new_nnz)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, &h_csrColIndA[h_csrRowPtrA[rank*(m/multi+1)]], (new_nnz)*sizeof(int),hipMemcpyHostToDevice);
    if( rank==multi-1 ) {
        hipMemcpy(d_csrRowPtrA, &h_csrRowPtrA[rank*new_n], (m-rank*new_n+1)*sizeof(int),hipMemcpyHostToDevice);
    } else {
        hipMemcpy(d_csrRowPtrA, &h_csrRowPtrA[rank*new_n], (new_n+1)*sizeof(int),hipMemcpyHostToDevice);
    }

    // Test copy data from device to host
    /*typeVal *h_csrValTest = (typeVal*)malloc(nnz*sizeof(typeVal));
    int *h_csrColIndTest = (int*)malloc(nnz*sizeof(int));
    int *h_csrRowPtrTest = (int*)malloc((m+1)*sizeof(int));
    int *h_rank = (int*)malloc(multi*sizeof(int));
    int *h_displs = (int*)malloc(multi*sizeof(int));
    //for( int i=0; i<multi; i++ ) h_displs[i] = h_csrRowPtrA[i*(m/multi+1)];
    for( int i=0; i<multi; i++ ) h_displs[i] = i*new_n;

    typeVal *d_csrValTest;
    int *d_csrRowPtrTest;
    int *d_csrColIndTest;
    int *d_rank;
    int *d_new_nnz;
    hipMalloc(&d_new_nnz, sizeof(int));
    hipMalloc(&d_rank, multi*sizeof(int));
    hipMalloc(&d_csrValTest, nnz*sizeof(typeVal));
    hipMalloc(&d_csrRowPtrTest, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndTest, nnz*sizeof(int));
    //hipMemcpy(d_new_nnz, &new_nnz, sizeof(int), hipMemcpyHostToDevice);
    printf("%d: %d col, %d nnz\n", rank, new_n, new_nnz);
    //MPI_Barrier(MPI_COMM_WORLD);
    
    //MPI_Gather( &new_nnz, 1, MPI_INT, h_rank, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Gather( &new_n, 1, MPI_INT, h_rank, 1, MPI_INT, 0, MPI_COMM_WORLD);
    //MPI_Barrier(MPI_COMM_WORLD);
    //if(rank==0)hipMemcpy(h_rank, d_rank, multi*sizeof(int), hipMemcpyDeviceToHost);
    if(rank==0)print_array(h_rank, multi);
    for( int i=0; i<multi; i++ ) {
        int valid_rank;
        if( i!=multi-1 ) valid_rank = h_csrRowPtrA[(i+1)*new_n]-h_csrRowPtrA[i*new_n];
        else valid_rank = nnz - h_csrRowPtrA[i*new_n];
        if(rank==0) printf("%d: %d\n", i, valid_rank);
        if( valid_rank != h_rank[i] && rank==0 ) printf("Error %d: %d != %d\n", i, valid_rank, h_rank[i]);
    }

    MPI_Gatherv(d_csrValA, new_nnz, MPI_FLOAT, d_csrValTest, h_rank, h_displs, MPI_FLOAT, 0, MPI_COMM_WORLD);
    MPI_Gatherv(d_csrRowPtrA, new_n, MPI_INT, d_csrRowPtrTest, h_rank, h_displs, MPI_INT, 0, MPI_COMM_WORLD); 
    //if(rank==0)hipMemcpy(h_csrValTest, d_csrValTest, nnz*sizeof(typeVal),hipMemcpyDeviceToHost);
    //if(rank==0)verify( nnz, h_csrValTest, h_csrValA );
    if(rank==0)hipMemcpy(h_csrRowPtrTest, d_csrRowPtrTest, m*sizeof(int),hipMemcpyDeviceToHost);
    if(rank==0)verify( m, h_csrRowPtrTest, h_csrRowPtrA );
    hipMemcpy(&h_csrColIndA[h_csrRowPtrA[rank*new_n]], d_csrColIndA, (new_nnz)*sizeof(int),hipMemcpyDeviceToHost);
    if( rank==multi-1 ) {
        hipMemcpy(&h_csrRowPtrA[rank*new_n], d_csrRowPtrA, (m-rank*new_n+1)*sizeof(int),hipMemcpyDeviceToHost);
    } else {
        hipMemcpy(&h_csrRowPtrA[rank*new_n], d_csrRowPtrA, (new_n+1)*sizeof(int),hipMemcpyDeviceToHost);
    }*/

    // 7. Run COO -> CSR kernel
    //coo2csr( d_cooRowIndA, new_nnz, new_n, d_csrRowPtrA );

    // 8. Run BFS on CPU. Need data in CSR form first.
    int depth = 1000;
    depth = bfsCPU( source, m, h_csrRowPtrA, h_csrColIndA, h_bfsResultCPU, depth );

    // 9. Run CSR -> CSC kernel
    //csr2csc<typeVal>( new_n, new_nnz, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_cscValA, d_cscRowIndA, d_cscColPtrA );

    // 10. Run BFS kernel on GPU
    // Experiment 1: Optimized BFS using mXv (no Val array)
    //spmspvBfs( source, nnz, m, h_csrRowPtrA, d_csrRowPtrA, d_csrColIndA, d_bfsResult, depth, *context); 

    // Experiment 2: Optimized BFS using mXv
    //bfsSparse( source, new_nnz, new_n, m, multi, rank, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_bfsResult, depth, *context); 
    // Compare with CPU BFS for errors
    int *h_bfsResultSmall = (int*)malloc(new_n*sizeof(int));
    hipMemcpy(h_bfsResultSmall,d_bfsResult,m*sizeof(int),hipMemcpyDeviceToHost);
    int *h_rank = (int*)malloc(multi*sizeof(int));
    int *h_displs = (int*)malloc(multi*sizeof(int));
    for( int i=0; i<multi; i++ ) h_displs[i] = i*new_n;
    MPI_Gather( &new_n, 1, MPI_INT, h_rank, 1, MPI_INT, 0, MPI_COMM_WORLD);
    
    MPI_Gatherv(h_bfsResultSmall, new_n, MPI_INT, h_bfsResult, h_rank, h_displs, MPI_INT, 0, MPI_COMM_WORLD); 
    verify( m, h_bfsResult, h_bfsResultCPU );
    //print_array(h_bfsResult, m);



    /*hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);

    hipFree(d_cscValA);
    hipFree(d_cscRowIndA);
    hipFree(d_cscColPtrA);
    hipFree(d_bfsResult);

    free(h_csrValA);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooValA);
    free(h_cooRowIndA);
    free(h_cooColIndA);
    free(h_bfsResult);
    free(h_bfsResultCPU);*/

    MPI_Finalize();
}

int main(int argc, char**argv) {
    runBfs(argc, argv);
}    
