#include "hip/hip_runtime.h"
// Puts everything together
// For now, just run V times.
// Optimizations: 
// -come up with good stopping criteria [done]
// -start from i=1 [done]
// -test whether float really are faster than ints
// -distributed idea
// -change nthread [done - doesn't work]
 
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <deque>
#include <hipsparse.h>
#include <moderngpu.cuh>

#include <util.cuh>
#include <spgemm.cuh>

#include <testBfs.cpp>
#include <string.h>
#include <fstream>

// Counts number of nnz in 1D partition
void histogramHorz( const int *h_cscColPtrA, const int *h_cscRowIndA, const int m, const int part_size )
{
	std::ofstream outf;
	outf.open("histogramA.csv", std::ofstream::out | std::ofstream::app);
	printf("Partition size: %d\n", part_size);
	outf << "New matrix\n";
	int curr_ind = 0;
	int last_ind = 0;
	for( int i=part_size; i<=m-part_size; i+=part_size )
	{
		last_ind = curr_ind;
		curr_ind = h_cscColPtrA[i];
		outf << i/part_size-1 << " " << curr_ind-last_ind << "\n";
	}
}

// Counts number of nnz in 1D partition
void histogramVert( const int *h_cscColPtrA, const int *h_cscRowIndA, const int m, const int part_size )
{
	std::ofstream outf;
	outf.open("histogramB.csv", std::ofstream::out | std::ofstream::app);
	printf("Partition size: %d\n", part_size);
	outf << "New matrix\n";
	int curr_ind = 0;
	int last_ind = 0;
	for( int i=part_size; i<=m-part_size; i+=part_size )
	{
		last_ind = curr_ind;
		curr_ind = h_cscColPtrA[i];
		outf << i/part_size-1 << " " << curr_ind-last_ind << "\n";
	}
}

// Counts number of nnz in 2D partition
void histogramBlock( const int *h_cscColPtrA, const int *h_cscRowIndA, const int m, const int part_size )
{
	std::ofstream outf, outf2;
	outf2.open("histogramC.csv", std::ofstream::out | std::ofstream::app);
	outf2 << "New matrix\n";

	printf("Partition size: %d\n", part_size);
	int curr_ind = 0;
	int last_ind = 0;

    int block_size = (m+part_size-1)/part_size;
	int *block = (int*) malloc( block_size*block_size*sizeof(int));
	printf("Block size: %d\n", block_size);
	for( int i=0; i<block_size*block_size; i++ )
		block[i] = 0;

	for( int i=part_size; i<=m-part_size; i+=part_size )
	{
		last_ind = curr_ind;
		curr_ind = h_cscColPtrA[i];
		for( int j=last_ind; j< curr_ind; j++ ) 
		{
			//printf("%d ", (i/part_size-1)*block_size+h_cscRowIndA[j]/part_size ); 
			block[(i/part_size-1)*block_size+h_cscRowIndA[j]/part_size]++;
		}
	}

	for( int i=0; i<block_size*block_size; i++ )
	{
		if( block[i]!=0 )
			outf2 << i/block_size << " " << i%block_size << " " << block[i] << "\n";
	}
}

int countDiag( const int edge, const int *h_cscRowIndA, const int* h_cooColIndA ) {
    int count = 0;
    for( int i=0; i<edge; i++ )
        if( h_cscRowIndA[i]==h_cooColIndA[i] )
            count++;
    return count;
}

template< typename T >
void buildVal( const int edge, T *h_cscValA ) {
    for( int i=0; i<edge; i++ )
        h_cscValA[i] = 1.0;
}

template< typename T >
void buildLower( const int m, const int edge_B, const int *h_cscColPtrA, const int *h_cscRowIndA, const T *h_cscValA, int *h_cscColPtrB, int *h_cscRowIndB, T *h_cscValB, bool lower=true ) {

    int count = 0;
    h_cscColPtrB[0] = count;
    for( int i=0; i<m; i++ ) {
        for( int j=h_cscColPtrA[i]; j<h_cscColPtrA[i+1]; j++ ) {
            if( (lower==true && h_cscRowIndA[j] > i) || (lower==false && h_cscRowIndA[j] < i) ) {
                //printf("%d %d %d\n", i, j, count);
                h_cscRowIndB[count] = h_cscRowIndA[j];
                h_cscValB[count] = h_cscValA[j];
                count++;
            }
        }
        h_cscColPtrB[i+1] = count;
    }
}

long long squareDegree( const int m, int *h_cscColPtrA ) {
    long long sum = 0;
    long long deg = 0;
    for( int i=0; i<m; i++ ) {
        deg = h_cscColPtrA[i+1] - h_cscColPtrA[i];
        sum += deg*deg;
    }
    return sum;
}

int maxDegree( const int m, int *h_cscColPtrA ) {
    int max = 0;
    int deg = 0;
    for( int i=0; i<m; i++ ) {
        deg = h_cscColPtrA[i+1] - h_cscColPtrA[i];
        if( deg > max )
            max = deg;
    }
    return max;
}

void runBfs(int argc, char**argv) { 
    int m, n, edge;
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(0);

    // Define what filetype edge value should be stored
    typedef float typeVal;

    // File i/o
    // 1. Open file from command-line 
    // -source 1
    freopen(argv[1],"r",stdin);
    int source;
    int device;
    float delta;
    bool undirected = false;
	bool weighted = false;
    if( parseArgs( argc, argv, source, device, delta, undirected )==true ) {
        printf( "Usage: test apple.mtx -source 5\n");
        return;
    }
    //hipSetDevice(device);
    printf("Testing %s from source %d\n", argv[1], source);
    
    // 2. Reads in number of edges, number of nodes
    readEdge( m, n, edge, stdin );
    printf("Graph has %d nodes, %d edges\n", m, edge);
    if( undirected )
      edge=2*edge;

    // 3. Allocate memory depending on how many edges are present
    typeVal *h_cscValA, *h_cooValA;
    //int *h_csrRowPtrA, *h_csrColIndA;
    int *h_cscRowIndA, *h_cscColPtrA;
    int *h_cooRowIndA, *h_cooColIndA;
    int *h_bfsResult, *h_bfsResultCPU;

    //h_csrValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_cscValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    h_cooValA    = (typeVal*)malloc(edge*sizeof(typeVal));
    //h_csrRowPtrA = (int*)malloc((m+1)*sizeof(int));
    //h_csrColIndA = (int*)malloc(edge*sizeof(int));
    h_cscColPtrA = (int*)malloc((m+1)*sizeof(int));
    h_cscRowIndA = (int*)malloc(edge*sizeof(int));
    h_cooRowIndA = (int*)malloc(edge*sizeof(int));
    h_cooColIndA = (int*)malloc(edge*sizeof(int));
    h_bfsResult = (int*)malloc((m)*sizeof(int));
    h_bfsResultCPU = (int*)malloc((m)*sizeof(int));

    // 4. Read in graph from .mtx file
    // We are actually doing readMtx<typeVal>( edge, h_cooColIndA, h_cscRowIndA, h_cscValA );
    CpuTimer cpu_timerRead;
    CpuTimer cpu_timerMake;
    CpuTimer cpu_timerBuild;
    printf("Old edge #: %d\n", edge);
    cpu_timerRead.Start();
	if( undirected )
    	weighted = readMtx<typeVal>( edge/2, h_cooColIndA, h_cooRowIndA, h_cooValA );
	else
    	weighted = readMtx<typeVal>( edge, h_cooColIndA, h_cooRowIndA, h_cooValA );
    cpu_timerRead.Stop();
    if( !weighted )
		buildVal( edge, h_cscValA );
    cpu_timerMake.Start();
    if( undirected ) {
		edge = makeSymmetric( edge, h_cooColIndA, h_cooRowIndA, h_cooValA );
    	printf("\nUndirected graph has %d nodes, %d edges\n", m, edge);
	}
    print_matrixCOO( h_cooValA, h_cooRowIndA, h_cooColIndA, m, edge );
    cpu_timerMake.Stop();
    cpu_timerBuild.Start();
    // This function reads CSR or CSC by swapping h_cooColIndA and h_cooRowIndA
    if( undirected )
		buildMatrix<typeVal>( h_cscColPtrA, h_cscRowIndA, h_cscValA, m, edge, h_cooRowIndA, h_cooColIndA, h_cooValA );
	else
    	buildMatrix<typeVal>( h_cscColPtrA, h_cscRowIndA, h_cscValA, m, edge, h_cooColIndA, h_cooRowIndA, h_cooValA );
    cpu_timerBuild.Stop();
    float elapsedRead = cpu_timerRead.ElapsedMillis();
    float elapsedMake = cpu_timerMake.ElapsedMillis();
    float elapsedBuild= cpu_timerBuild.ElapsedMillis();
    printf("readMtx: %f ms\n", elapsedRead);
    printf("makeSym: %f ms\n", elapsedMake);
    printf("buildMat: %f ms\n", elapsedBuild);

    // 4b. Count diagonal
    int diag = countDiag( edge, h_cooRowIndA, h_cooColIndA ); 
    int edge_B = edge;
    int edge_C = edge_B;
    printf("Number of elements on diagonal: %d\n", diag);
    printf("Number of elements on L: %d\n", edge_B);
    printf("The max degree is: %d\n", maxDegree(m, h_cscColPtrA));
    printf("Square degree sum is: %lld\n", squareDegree(m, h_cscColPtrA));

    // 4c. Allocate memory to second and third matrices
    int m_B = m;
    int m_C = m;

    typeVal *h_cscValB;
    int *h_cscRowIndB, *h_cscColPtrB;
    h_cscValB = (typeVal*)malloc(edge_B*sizeof(typeVal));
    h_cscRowIndB = (int*)malloc(edge_B*sizeof(int));
    h_cscColPtrB = (int*)malloc((m_B+1)*sizeof(int));

    if( !weighted )
		buildVal( edge, h_cscValA );
    //buildLower( m, edge_B, h_cscColPtrA, h_cscRowIndA, h_cscValA, h_cscColPtrB, h_cscRowIndB, h_cscValB );
	memcpy( h_cscColPtrB, h_cscColPtrA, (m_B+1)*sizeof(int));
	memcpy( h_cscRowIndB, h_cscRowIndA, edge_B*sizeof(int));
	memcpy( h_cscValB, h_cscValA, edge_B*sizeof(typeVal));
    print_matrix( h_cscValA, h_cscColPtrA, h_cscRowIndA, m );
    print_matrix( h_cscValB, h_cscColPtrB, h_cscRowIndB, m );

    // 5. Allocate GPU memory
    typeVal *d_cscValA, *d_cscValB, *d_cscValC;
    //typeVal *d_csrValA;
    //int *d_csrRowPtrA, *d_csrColIndA;
    int *d_cscRowIndA, *d_cscColPtrA;
    int *d_cscRowIndB, *d_cscColPtrB;
    int *d_cscRowIndC, *d_cscColPtrC;
    int *d_cooColIndA;
    int *d_bfsResult;
    hipMalloc(&d_bfsResult, m*sizeof(int));

    //hipMalloc(&d_csrValA, edge*sizeof(typeVal));
    //hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    //hipMalloc(&d_csrColIndA, edge*sizeof(int));
    //hipMalloc(&d_cooRowIndA, edge*sizeof(int));
    //hipMalloc(&d_cooColIndA, edge*sizeof(int));
    hipMalloc(&d_cscValA, edge*sizeof(typeVal));
    hipMalloc(&d_cscRowIndA, edge*sizeof(int));
    hipMalloc(&d_cscColPtrA, (m+1)*sizeof(int));

    // 5b GPU memory for matrices B and C
    hipMalloc(&d_cscValB, edge_B*sizeof(typeVal));
    hipMalloc(&d_cscRowIndB, edge_B*sizeof(int));
    hipMalloc(&d_cscColPtrB, (m_B+1)*sizeof(int));

    // 6. Copy data from host to device
    hipMemcpy(d_cscValA, h_cscValA, (edge)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_cscRowIndA, h_cscRowIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cscColPtrA, h_cscColPtrA, (m+1)*sizeof(int),hipMemcpyHostToDevice);

    // 6b Copy data from host to device for matrices B and C
    hipMemcpy(d_cscValB, h_cscValB, (edge_B)*sizeof(typeVal),hipMemcpyHostToDevice);
    hipMemcpy(d_cscRowIndB, h_cscRowIndB, (edge_B)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cscColPtrB, h_cscColPtrB, (m+1)*sizeof(int),hipMemcpyHostToDevice);

    // 7. [insert CPU verification code here] 

    // 8. Make GPU timers
    GpuTimer gpu_timer;
    float elapsed = 0.0f;

    // 9. Run spgemm
    // Must be UxL because we are using CSC matrices rather than specified CSR
    // input required by cuSPARSE
    int NT = 512;
    int NB = (m+NT-1)/NT;
    gpu_timer.Start();
    
    //edge_D = mXm<typeVal>( edge, m, d_cscValA, d_cscColPtrA, d_cscRowIndA, d_cscValA, h_cscColPtrA, d_cscColPtrA, d_cscRowIndA, d_cscValD, d_cscColPtrD, d_cscRowIndD, *context);
    edge_C = spgemm<typeVal>( edge, m, d_cscValA, d_cscColPtrA, d_cscRowIndA, d_cscValB, d_cscColPtrB, d_cscRowIndB, d_cscValC, d_cscColPtrC, d_cscRowIndC );
    gpu_timer.Stop();
    elapsed += gpu_timer.ElapsedMillis();
    printf("spgemm finished in %f msec.\n", elapsed);

    // 10. Allocate memory for C on Host
	typeVal *h_cscValC;
    int *h_cscRowIndC, *h_cscColPtrC;
    h_cscValC = (typeVal*)malloc(edge_C*sizeof(typeVal));
    h_cscRowIndC = (int*)malloc(edge_C*sizeof(int));
    h_cscColPtrC = (int*)malloc((m_C+1)*sizeof(int));

	// Statistics:
	// MEMORY = 128000 (L2), 1000 (L1)
	typeVal *h_cscValD;
	int *h_cscRowIndD, *h_cscColPtrD;
    h_cscValD = (typeVal*)malloc(edge*sizeof(typeVal));
    h_cscRowIndD = (int*)malloc(edge*sizeof(int));
    h_cscColPtrD = (int*)malloc((m+1)*sizeof(int));
    buildMatrix<typeVal>( h_cscColPtrD, h_cscRowIndD, h_cscValD, m, edge, h_cooRowIndA, h_cooColIndA, h_cooValA );
	float AGGRO_FACTOR = 0.5;  // a value in (0-1] that describes how close to 
								// shared mem threshold
	float k_A = (float)edge/m;
	float MEMORY = 128000.0;
	float TARGET_PART_SIZE = AGGRO_FACTOR*MEMORY/k_A;
	float TARGET_PART_NUM = edge/MEMORY/AGGRO_FACTOR;

	printf("Mem: %f; Size: %f; Num: %f\n", MEMORY, (int)TARGET_PART_SIZE, TARGET_PART_NUM);

    // 11. Compare with CPU BFS for errors
    hipMemcpy(h_cscRowIndC, d_cscRowIndC, edge_C*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_cscColPtrC, d_cscColPtrC, (m_C+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_cscValC, d_cscValC, edge_C*sizeof(typeVal), hipMemcpyDeviceToHost);
    //printf("Matrix C: %dx%d with %d nnz\n", m, m, edge_C);
    //print_matrix( h_cscValC, h_cscColPtrC, h_cscRowIndC, m );

    histogramHorz( h_cscColPtrA, h_cscRowIndA, m, (int)TARGET_PART_SIZE );
    histogramVert( h_cscColPtrD, h_cscRowIndD, m, (int)TARGET_PART_SIZE );

	histogramBlock( h_cscColPtrC, h_cscRowIndC, m, (int)TARGET_PART_SIZE );
}

int main(int argc, char**argv) {
    runBfs(argc, argv);
}    
