// Puts everything together

#include <cstdlib>
#include <ctime>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

/*void read_mtx( char** filename ) {
    freopen(filename,"r",stdin);
    //freopen("sample.out","w",stdout);

    scanf("%d", &
}*/

void print_array( int* array, int length ) {
    for( int j=0;j<length;j++ ) {
        printf("%d %d\n", array[j], j);
    }
}

int main(int argc, char** argv) {
    int m, n, edge;

    //for( int i=1; i<argc; i++ ) {
    //bfs(argv[i]);
    freopen(argv[1],"r",stdin);
    //freopen("sample.out","w",stdout);

    scanf("%d %d %d", &m, &n, &edge);
    
    // Allocate memory depending on how many edges are present
    float *h_csrValA;
    int *h_csrRowPtrA, *h_csrColIndA, *h_cooRowIndA;

    h_csrValA    = (float *)malloc(edge*sizeof(float));
    h_csrRowPtrA = (int *)malloc((m+1)*sizeof(int));
    h_csrColIndA = (int *)malloc(edge*sizeof(int));
    h_cooRowIndA = (int *)malloc(edge*sizeof(int));

    for( int j=0; j<edge; j++ ) {
        if( scanf("%d", &h_csrColIndA[j])==EOF ) {
            printf("Error: not enough rows in mtx file.\n");
            break;
        }
        scanf("%d", &h_cooRowIndA[j]);
        h_csrValA[j]=1.0;
        printf("%d %d %d\n", h_cooRowIndA[j], h_csrColIndA[j], j);
    }

    // Allocate GPU memory
    float *d_csrValA;
    int *d_csrRowPtrA, *d_csrColIndA, *d_cooRowIndA;

    hipMalloc(&d_csrValA, edge*sizeof(float));
    hipMalloc(&d_csrRowPtrA, (m+1)*sizeof(int));
    hipMalloc(&d_csrColIndA, edge*sizeof(int));
    hipMalloc(&d_cooRowIndA, edge*sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_csrValA, h_csrValA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_cooRowIndA, h_cooRowIndA, (edge)*sizeof(int),hipMemcpyHostToDevice);
    //cudaMemcpy(h_cooRowIndA, d_cooRowIndA, (edge)*sizeof(int),cudaMemcpyDeviceToHost);
    //print_array(h_cooRowIndA,edge);

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // Convert from COO -> CSR
    hipsparseStatus_t status = hipsparseXcoo2csr(handle, d_cooRowIndA, edge, m, d_csrRowPtrA, HIPSPARSE_INDEX_BASE_ZERO);
    switch( status ) {
        case HIPSPARSE_STATUS_SUCCESS:
            printf("COO -> CSR conversion successful!\n");
            break;
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            printf("Error: Library not initialized.\n");
            break;
        case HIPSPARSE_STATUS_INVALID_VALUE:
            printf("Error: Invalid value for idxbase.\n");
            break;
        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            printf("Error: Failed to launch GPU.\n");
    }

    // BFS

    // Copy data back to host
    hipMemcpy(h_csrRowPtrA,d_csrRowPtrA,(m+1)*sizeof(int),hipMemcpyDeviceToHost);
    print_array(h_csrRowPtrA,m+1);

    hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_cooRowIndA);

    free(h_csrValA);
    free(h_csrRowPtrA);
    free(h_csrColIndA);
    free(h_cooRowIndA);
}
